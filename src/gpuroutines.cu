#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"
#include "hipsolver.h"

#define BLOCK_SIZE 1024
#define TILE_DIM 32 
#define BLOCK_ROWS 8

__global__ void CaddKernel(hipComplex* c, const hipComplex alpha, const hipComplex* a, const hipComplex beta, const hipComplex* b, int size)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < size) {
      c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) + (beta.x * b[i].x - beta.y * b[i].y);
      c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) + (beta.x * b[i].y + beta.y * b[i].x);
   }
}

__global__ void ZaddKernel(hipDoubleComplex* c, const hipDoubleComplex alpha, const hipDoubleComplex* a, const hipDoubleComplex beta, const hipDoubleComplex* b, int size)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < size) {
      c[i].x = (alpha.x * a[i].x - alpha.y * a[i].y) + (beta.x * b[i].x - beta.y * b[i].y);
      c[i].y = (alpha.x * a[i].y + alpha.y * a[i].x) + (beta.x * b[i].y + beta.y * b[i].x);
   }
}

/*
__global__ void hermitian(cuComplex *odata, const cuComplex *idata)
{
  __shared__ cuComplex tile[TILE_DIM][TILE_DIM];
  
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {	   
     odata[(y+j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x;
     odata[(y+j)*width + x].y = -tile[threadIdx.x][threadIdx.y + j].y;
  }
}
*/
__global__ void CinitKernel(hipComplex *a, int nrow) {
    
    int size;	 
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
              a[i].x = 1.0;
              a[i].y = 0.0;
	     }
          else{
              a[i].x = 0.0;
              a[i].y = 0.0;
	   }
    }
}

__global__ void ZinitKernel(hipDoubleComplex *a, int nrow) {
    
    int size;	 
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
              a[i].x = 1.0;
              a[i].y = 0.0;
	     }
          else{
              a[i].x = 0.0;
              a[i].y = 0.0;
	   }
    }
}

__global__ void CtraceKernel(hipComplex *a, int nrow, hipComplex *trace) {
    
    int size;	 
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
             trace[i].x = a[i].x;
             trace[i].y = 0.0;
             }
          else{
             trace[i].x = 0.0;
             trace[i].y = 0.0;
	     }
    }
}

__global__ void ZtraceKernel(hipDoubleComplex *a, int nrow, hipDoubleComplex *trace) {
    
    int size;	 
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    size = nrow*nrow;
    if(i < size) {
          if(i%(nrow+1) == 0){
             trace[i].x = a[i].x;
             trace[i].y = 0.0;
             }
          else{
             trace[i].x = 0.0;
             trace[i].y = 0.0;
	     }
    }
}
/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ DATA MOVEMENT  ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_createMat(void **d_A, int bytecount)
{
  hipError_t err;
  err = hipMalloc(d_A, bytecount);
  printf("GPU Address: %p \n",*d_A);
  return err;
}

extern "C" int cu_copyMatH2D(void *h_A, void *d_A, int bytecount)
{
  hipError_t err;
  printf("copy %p to %p\n",h_A,d_A);
  err = hipMemcpy(d_A, h_A, bytecount, hipMemcpyHostToDevice);  
  return err;
}	

extern "C" int cu_copyMatD2H(void *h_A, void *d_A, int bytecount)
{
  hipError_t err;
  printf("copy %p to %p\n",d_A,h_A);
  err = hipMemcpy(h_A, d_A, bytecount, hipMemcpyDeviceToHost);  
  return err;
}	

extern "C" int cu_deleteMat(void *d_A)
{
  hipError_t err;
  printf("add_free: %p",d_A);
  err = hipFree(d_A);  
  return err;
}	

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ INIT/FINAL ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_cublasInit(hipblasHandle_t *hcublas)
{
  hipblasStatus_t err;
  err = hipblasCreate(hcublas);
  if (err != 0){
    printf("cublas create error: %d\n",err);
  }
  printf("hcublas Addr: %p \n",*hcublas);
  return err;
}

extern "C" int cu_cublasFinalize(hipblasHandle_t hcublas)
{
  hipblasStatus_t err;
  err = hipblasDestroy(hcublas); 
  return err;
}

extern "C" int cu_cusolverInit(hipsolverHandle_t *hcusolver)
{
  hipsolverStatus_t err;
  err = hipsolverDnCreate(hcusolver);
  if (err != 0){
    printf("cusolver create error: %d\n",err);
  }
  printf("hcusolver Addr: %p \n",*hcusolver);
  return err;
}

extern "C" int cu_cusolverFinalize(hipsolverHandle_t hcusolver)
{
  hipsolverStatus_t err;
  err = hipsolverDnDestroy(hcusolver); 
  return err;
}

/*~-~-~-~-~-~-~-~-~-~-~-~-~-~ MATRIX ROUTINES -~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-~-*/

extern "C" int cu_CmultMat(hipblasHandle_t hcublas, int m, int n, int k, hipComplex *alpha, void *d_A, void *d_B, hipComplex *beta, void *d_C, int dagger)
{
  hipComplex *pdA, *pdB, *pdC; 	

  printf("A: %p B: %p C: %p\n",d_A,d_B,d_C);
  pdA=(hipComplex *) d_A;
  pdB=(hipComplex *) d_B;
  pdC=(hipComplex *) d_C;
  hipblasStatus_t err;
  if (dagger == 0){
     err = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k, beta, pdC, m);
  }
  if (dagger == 1){
     err = hipblasCgemm(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k, beta, pdC, m);
  }
  if (dagger == 2){  
     err = hipblasCgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n, beta, pdC, m);
  }
  return err;
}

extern "C" int cu_ZmultMat(hipblasHandle_t hcublas, int m, int n, int k, hipDoubleComplex *alpha, void *d_A, void *d_B, hipDoubleComplex *beta, void *d_C, int dagger)
{
  hipDoubleComplex *pdA, *pdB, *pdC; 	

  printf("A: %p B: %p C: %p\n",d_A,d_B,d_C);
  pdA=(hipDoubleComplex *) d_A;
  pdB=(hipDoubleComplex *) d_B;
  pdC=(hipDoubleComplex *) d_C;
  hipblasStatus_t err;
  if (dagger == 0){
     err = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, pdA, m, pdB, k, beta, pdC, m);
  }
  if (dagger == 1){
     err = hipblasZgemm(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, k, alpha, pdA, k, pdB, k, beta, pdC, m);
  }
  if (dagger == 2){  
     err = hipblasZgemm(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, k, alpha, pdA, m, pdB, n, beta, pdC, m);
  }
  return err;
}

extern "C" int cu_Cinverse(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A, void *d_Ainv, int N)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t;
   // declare arrays on the device
   hipComplex  *pdA , *pdAinv, *d_LU, *d_Work; 
  
   pdA = (hipComplex *) d_A;
   pdAinv = (hipComplex *) d_Ainv;
   // coeff . matrix , rhs , workspace
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int info_gpu = 0;

   // compute buffer size and prep . memory
   cusolverStatus = hipsolverDnCgetrf_bufferSize( hcusolver, N , N , pdA , N , &Lwork);
   // prepare memory on the device

   cudaStatus = hipMalloc(( void **)& d_LU, N*N*sizeof(hipComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot , N*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_info , sizeof(int));
   // copy d_LU <- pdA
   hipblasStatus_t = hipblasCcopy(hcublas, N*N, pdA, 1, d_LU, 1);

   cudaStatus = hipMalloc(( void **)& d_Work , Lwork*sizeof(hipComplex));
  
   // LU factorization of d_A , with partial pivoting and row
   // interchanges ; row i is interchanged with row d_pivot ( i );
   cusolverStatus = hipsolverDnCgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);
  
   // use the LU factorization to solve the system d_LU * x = d_Ainv ;
   // the solution overwrites d_Ainv
   cusolverStatus = hipsolverDnCgetrs(hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info);

   cudaStatus = hipMemcpy(&info_gpu , d_info , sizeof(int), hipMemcpyDeviceToHost);
   // d_info -> info_gpu
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_Work);
   cudaStatus = hipFree(d_LU);
   return cudaStatus;
}

extern "C" int cu_Zinverse(hipblasHandle_t hcublas, hipsolverHandle_t hcusolver, void *d_A, void *d_Ainv, int N)
{
   hipError_t cudaStatus;
   hipsolverStatus_t cusolverStatus;
   hipblasStatus_t hipblasStatus_t; 
   // declare arrays on the device
   hipDoubleComplex  *pdA , *pdAinv, *d_LU, *d_Work; 
  
   pdA = (hipDoubleComplex *) d_A;
   pdAinv = (hipDoubleComplex *) d_Ainv;
   // coeff . matrix , rhs , workspace
   int *d_pivot , *d_info , Lwork ; // pivots , info , worksp . size
   int info_gpu = 0;

   // compute buffer size and prep . memory
   cusolverStatus = hipsolverDnZgetrf_bufferSize( hcusolver, N , N , pdA , N , &Lwork);
   // prepare memory on the device

   cudaStatus = hipMalloc(( void **)& d_LU, N*N*sizeof(hipDoubleComplex));
   cudaStatus = hipMalloc(( void **)& d_pivot , N*sizeof(int));
   cudaStatus = hipMalloc(( void **)& d_info , sizeof(int));
   // copy d_LU <- pdA
   hipblasStatus_t = hipblasZcopy(hcublas, N*N, pdA, 1, d_LU, 1);

   cudaStatus = hipMalloc(( void **)& d_Work , Lwork*sizeof(hipDoubleComplex));
  
   // LU factorization of d_A , with partial pivoting and row
   // interchanges ; row i is interchanged with row d_pivot ( i );
   cusolverStatus = hipsolverDnZgetrf(hcusolver, N, N, d_LU, N, d_Work, d_pivot, d_info);
  
   // use the LU factorization to solve the system d_LU * x = d_Ainv ;
   // the solution overwrites d_Ainv
   cusolverStatus = hipsolverDnZgetrs(hcusolver, HIPBLAS_OP_N, N, N, d_LU, N, d_pivot, pdAinv, N, d_info);

   cudaStatus = hipMemcpy(&info_gpu , d_info , sizeof(int), hipMemcpyDeviceToHost);
   // d_info -> info_gpu
   cudaStatus = hipFree(d_pivot);
   cudaStatus = hipFree(d_info);
   cudaStatus = hipFree(d_Work);
   cudaStatus = hipFree(d_LU);
   return cudaStatus;
}

extern "C" int cu_Ckernelsum(void *d_C, hipComplex *alpha, void *d_A, hipComplex *beta, void *d_B, int size)
{
   int NumBlocks;
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;
   hipComplex *pdC = (hipComplex *) d_C;

   NumBlocks = (size/BLOCK_SIZE)+1;

   CaddKernel<<<NumBlocks,BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

   return 0; 
}

extern "C" int cu_Zkernelsum(void *d_C, hipDoubleComplex *alpha, void *d_A, hipDoubleComplex *beta, void *d_B, int size)
{
   int NumBlocks;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;
   hipDoubleComplex *pdC = (hipDoubleComplex *) d_C;

   NumBlocks = (size/BLOCK_SIZE)+1;

   ZaddKernel<<<NumBlocks,BLOCK_SIZE>>>(pdC, *alpha, pdA, *beta, pdB, size);

   return 0; 
}

extern "C" int cu_Cmatsum(hipblasHandle_t hcublas, int m, int n, hipComplex *alpha, void *d_A, hipComplex *beta, void *d_B, void *d_C, int dagger)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C  	
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;
   hipComplex *pdC = (hipComplex *) d_C;

   hipblasStatus_t err;
   if (dagger == 0) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB, m, pdC, m);
      }
   if (dagger == 1) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB, m, pdC, m);
      }
   if (dagger == 2) {
      err = hipblasCgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB, n, pdC, m);
      }
   return err; 
}

extern "C" int cu_Zmatsum(hipblasHandle_t hcublas, int m, int n, hipDoubleComplex *alpha, void *d_A, hipDoubleComplex *beta, void *d_B, void *d_C, int dagger)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C  	
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;
   hipDoubleComplex *pdC = (hipDoubleComplex *) d_C;

   hipblasStatus_t err;
   if (dagger == 0) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, alpha, pdA, m, beta, pdB, m, pdC, m);
      }
   if (dagger == 1) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_C, HIPBLAS_OP_N, m, n, alpha, pdA, n, beta, pdB, m, pdC, m);
      }
   if (dagger == 2) {
      err = hipblasZgeam(hcublas, HIPBLAS_OP_N, HIPBLAS_OP_C, m, n, alpha, pdA, m, beta, pdB, n, pdC, m);
      }
   return err; 
}

extern "C" int cu_Cinitmat( void *d_A, int nrow)
{
   int NumBlocks;
   int size = nrow*nrow;
   hipComplex *pdA = (hipComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;

   CinitKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow);

   return 0; 
}

extern "C" int cu_Zinitmat( void *d_A, int nrow)
{
   int NumBlocks;
   int size = nrow*nrow;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;

   ZinitKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow);

   return 0; 
}

extern "C" float cu_Ctrace(hipblasHandle_t hcublas, void *d_A, int nrow)
{
   hipError_t cudaStatus;
   hipblasStatus_t err;
   
   int NumBlocks;
   float result;
   int size = nrow*nrow;
   hipComplex *d_work;
   hipComplex *pdA = (hipComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;
   
   cudaStatus = hipMalloc(( void **)& d_work, size*sizeof(hipComplex));

   CtraceKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow, d_work);
   err = hipblasScasum(hcublas, size, d_work, 1, &result);

   cudaStatus = hipFree(d_work);
   
   return result;
}

extern "C" double cu_Ztrace(hipblasHandle_t hcublas, void *d_A, int nrow)
{
   hipError_t cudaStatus;
   hipblasStatus_t err;
   
   int NumBlocks;
   double result;
   int size = nrow*nrow;
   hipDoubleComplex *d_work;
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;

   NumBlocks = (size/BLOCK_SIZE)+1;
   
   cudaStatus = hipMalloc(( void **)& d_work, size*sizeof(hipDoubleComplex));

   ZtraceKernel<<<NumBlocks,BLOCK_SIZE>>>(pdA, nrow, d_work);
   err = hipblasDzasum(hcublas, size, d_work, 1, &result);

   cudaStatus = hipFree(d_work);
   
   return result;
}

extern "C" int cu_Cmatcopy(hipblasHandle_t hcublas,  void *d_A,  void *d_B, int N)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C  	
   hipComplex *pdA = (hipComplex *) d_A;
   hipComplex *pdB = (hipComplex *) d_B;

   hipblasStatus_t err;
   
   err = hipblasCcopy(hcublas, N*N, pdA, 1, pdB, 1);
   return err; 
}

extern "C" int cu_Zmatcopy(hipblasHandle_t hcublas,  void *d_A,  void *d_B, int size)
{
   //m number of rows of matrix op(A) and C
   //n number of columns of matrix op(B) and C  	
   hipDoubleComplex *pdA = (hipDoubleComplex *) d_A;
   hipDoubleComplex *pdB = (hipDoubleComplex *) d_B;

   hipblasStatus_t err;
   
   err = hipblasZcopy(hcublas, size, pdA, 1, pdB, 1);
   return err; 
}

